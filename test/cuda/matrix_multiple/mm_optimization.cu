
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <random>
#include <stdexcept>
#include <vector>

#define BLOCK_DIM 32

#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <typename T>
std::vector<T> create_rand_vector(size_t n)
{
    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(-256, 256);

    std::vector<T> vec(n);
    for (size_t i{0}; i < n; ++i)
    {
        vec.at(i) = static_cast<T>(uniform_dist(e));
    }

    return vec;
}

// mat_1: m x n
// mat_2: n x p
// mat_3: m x p
template <typename T>
void mm(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n, size_t p)
{
    // Compute the cells in mat_3 sequentially.
    for (size_t i{0}; i < m; ++i)
    {
        for (size_t j{0}; j < p; ++j)
        {
            T acc_sum{0};
            for (size_t k{0}; k < n; ++k)
            {
                acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
            }
            mat_3[i * p + j] = acc_sum;
        }
    }
}

template <typename T>
__global__ void mm_kernel(T const* mat_1, T const* mat_2, T* mat_3, size_t m,
                          size_t n, size_t p)
{
    // 2D block and 2D thread
    // Each thread computes one cell in mat_3.
    size_t i{blockIdx.y * blockDim.y + threadIdx.y};
    size_t j{blockIdx.x * blockDim.x + threadIdx.x};

    // Do not process outside the matrix.
    // Do not forget the equal sign!
    if ((i >= m) || (j >= p))
    {
        return;
    }

    T acc_sum{0};
    for (size_t k{0}; k < n; ++k)
    {
        acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
    }
    mat_3[i * p + j] = acc_sum;
}

template <typename T>
__global__ void mm_kernel_optimized(T const* mat_1, T const* mat_2, T* mat_3,
                                    size_t m, size_t n, size_t p)
{
    __shared__ T mat_1_tile[BLOCK_DIM][BLOCK_DIM];
    __shared__ T mat_2_tile[BLOCK_DIM][BLOCK_DIM];

    T acc_sum{0};

    for (size_t tile_idx{0};
         tile_idx < ceilf(static_cast<float>(n) / BLOCK_DIM); ++tile_idx)
    {
        size_t i{blockIdx.y * blockDim.y + threadIdx.y};
        size_t j{tile_idx * blockDim.x + threadIdx.x};
        if ((i < m) && (j < n))
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = mat_1[i * n + j];
        }
        else
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = 0;
        }
        i = tile_idx * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        if ((i < n) && (j < p))
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = mat_2[i * p + j];
        }
        else
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        for (size_t k{0}; k < BLOCK_DIM; ++k)
        {
            acc_sum += mat_1_tile[threadIdx.y][k] * mat_2_tile[k][threadIdx.x];
        }
        __syncthreads();
    }

    // 2D block and 2D thread
    // Each thread computes one cell in mat_3.
    size_t i{blockIdx.y * blockDim.y + threadIdx.y};
    size_t j{blockIdx.x * blockDim.x + threadIdx.x};

    if ((i < m) && (j < p))
    {
        mat_3[i * p + j] = acc_sum;
    }
}

template <typename T>
void mm_cuda(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n,
             size_t p,
             void (*f)(T const*, T const*, T*, size_t, size_t, size_t))
{
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks_per_grid(1, 1);
    blocks_per_grid.x = std::ceil(static_cast<double>(p) /
                                  static_cast<double>(threads_per_block.x));
    blocks_per_grid.y = std::ceil(static_cast<double>(m) /
                                  static_cast<double>(threads_per_block.y));
    f<<<blocks_per_grid, threads_per_block>>>(mat_1, mat_2, mat_3, m, n, p);
}

template <typename T>
bool allclose(std::vector<T> const& vec_1, std::vector<T> const& vec_2,
              T const& abs_tol)
{
    if (vec_1.size() != vec_2.size())
    {
        return false;
    }
    for (size_t i{0}; i < vec_1.size(); ++i)
    {
        if (std::abs(vec_1.at(i) - vec_2.at(i)) > abs_tol)
        {
            std::cout << vec_1.at(i) << " " << vec_2.at(i) << std::endl;
            return false;
        }
    }
    return true;
}

template <typename T>
bool random_test_mm_cuda(size_t m, size_t n, size_t p,
                         void (*f)(T const*, T const*, T*, size_t, size_t,
                                   size_t))
{
    std::vector<T> const mat_1_vec{create_rand_vector<T>(m * n)};
    std::vector<T> const mat_2_vec{create_rand_vector<T>(n * p)};
    std::vector<T> mat_3_vec(m * p);
    std::vector<T> mat_4_vec(m * p);
    T const* mat_1{mat_1_vec.data()};
    T const* mat_2{mat_2_vec.data()};
    T* mat_3{mat_3_vec.data()};
    T* mat_4{mat_4_vec.data()};

    mm(mat_1, mat_2, mat_3, m, n, p);

    T *d_mat_1, *d_mat_2, *d_mat_4;

    // Allocate device buffer.
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * mat_1_vec.size()));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * mat_2_vec.size()));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * mat_4_vec.size()));

    // Copy data from host to device.
    checkCuda(hipMemcpy(d_mat_1, mat_1, sizeof(T) * mat_1_vec.size(),
                         hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_mat_2, mat_2, sizeof(T) * mat_2_vec.size(),
                         hipMemcpyHostToDevice));

    // Run matrix multiplication on GPU.
    mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p, f);
    hipDeviceSynchronize();
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
    // Copy data from device to host.
    checkCuda(hipMemcpy(mat_4, d_mat_4, sizeof(T) * mat_4_vec.size(),
                         hipMemcpyDeviceToHost));

    // Free device buffer.
    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    return allclose<T>(mat_3_vec, mat_4_vec, 1e-4);
}

template <typename T>
bool random_multiple_test_mm_cuda(size_t num_tests,
                                  void (*f)(T const*, T const*, T*, size_t,
                                            size_t, size_t))
{
    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(1, 256);

    size_t m{0}, n{0}, p{0};
    bool success{false};

    for (size_t i{0}; i < num_tests; ++i)
    {
        m = static_cast<size_t>(uniform_dist(e));
        n = static_cast<size_t>(uniform_dist(e));
        p = static_cast<size_t>(uniform_dist(e));
        success = random_test_mm_cuda<T>(m, n, p, f);
        if (!success)
        {
            return false;
        }
    }

    return true;
}

template <typename T>
float measure_latency_mm_cuda(size_t m, size_t n, size_t p, size_t num_tests,
                              size_t num_warmups,
                              void (*f)(T const*, T const*, T*, size_t, size_t,
                                        size_t))
{
    hipEvent_t startEvent, stopEvent;
    float time{0.0f};

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    T *d_mat_1, *d_mat_2, *d_mat_4;

    // Allocate device buffer.
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * m * n));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * n * p));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * m * p));

    for (size_t i{0}; i < num_warmups; ++i)
    {
        mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p, f);
    }

    checkCuda(hipEventRecord(startEvent, 0));
    for (size_t i{0}; i < num_tests; ++i)
    {
        mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p, f);
    }
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
    checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));

    // Free device buffer.
    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    float latency{time / num_tests};

    return latency;
}

int main()
{
    constexpr size_t num_tests{10};

    assert(random_multiple_test_mm_cuda<int32_t>(num_tests, mm_kernel));
    assert(random_multiple_test_mm_cuda<float>(num_tests, mm_kernel));
    assert(random_multiple_test_mm_cuda<double>(num_tests, mm_kernel));

    assert(
        random_multiple_test_mm_cuda<int32_t>(num_tests, mm_kernel_optimized));
    assert(random_multiple_test_mm_cuda<float>(num_tests, mm_kernel_optimized));
    assert(
        random_multiple_test_mm_cuda<double>(num_tests, mm_kernel_optimized));

    constexpr size_t num_measurement_tests{100};
    constexpr size_t num_measurement_warmups{10};
    const size_t m{1024}, n{1024}, p{1024};

    float mm_cuda_int32_latency{measure_latency_mm_cuda<int32_t>(
        m, n, p, num_measurement_tests, num_measurement_warmups, mm_kernel)};
    float mm_cuda_float_latency{measure_latency_mm_cuda<float>(
        m, n, p, num_measurement_tests, num_measurement_warmups, mm_kernel)};
    float mm_cuda_double_latency{measure_latency_mm_cuda<double>(
        m, n, p, num_measurement_tests, num_measurement_warmups, mm_kernel)};

    std::cout << "Matrix Multiplication CUDA Latency" << std::endl;
    std::cout << "m: " << m << " "
              << "n: " << n << " "
              << "p: " << p << std::endl;
    std::cout << "INT32: " << std::fixed << std::setprecision(5)
              << mm_cuda_int32_latency << " ms" << std::endl;
    std::cout << "FLOAT: " << std::fixed << std::setprecision(5)
              << mm_cuda_float_latency << " ms" << std::endl;
    std::cout << "DOUBLE: " << std::fixed << std::setprecision(5)
              << mm_cuda_double_latency << " ms" << std::endl;

    mm_cuda_int32_latency = measure_latency_mm_cuda<int32_t>(
        m, n, p, num_measurement_tests, num_measurement_warmups,
        mm_kernel_optimized);
    mm_cuda_float_latency = measure_latency_mm_cuda<float>(
        m, n, p, num_measurement_tests, num_measurement_warmups,
        mm_kernel_optimized);
    mm_cuda_double_latency = measure_latency_mm_cuda<double>(
        m, n, p, num_measurement_tests, num_measurement_warmups,
        mm_kernel_optimized);

    std::cout << "Optimized Matrix Multiplication CUDA Latency" << std::endl;
    std::cout << "m: " << m << " "
              << "n: " << n << " "
              << "p: " << p << std::endl;
    std::cout << "INT32: " << std::fixed << std::setprecision(5)
              << mm_cuda_int32_latency << " ms" << std::endl;
    std::cout << "FLOAT: " << std::fixed << std::setprecision(5)
              << mm_cuda_float_latency << " ms" << std::endl;
    std::cout << "DOUBLE: " << std::fixed << std::setprecision(5)
              << mm_cuda_double_latency << " ms" << std::endl;
}